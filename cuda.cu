#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "bricksorter.h"

extern "C" __global__
void CudaKernel(void)
{
  printf("hello world from cuda <%d,%d>\n",blockIdx.x, threadIdx.x);
  return;
}
extern "C" void CudaRunCodeFromC(void)
{
  CudaKernel<<<10, 1>>>();
  hipDeviceSynchronize();
  return;
}

extern "C" __global__
void BrickSortKernel(void)
{
  printf("hello from cuda brick sorter <%d,%d>\n",blockIdx.x, threadIdx.x);
  return;
}
extern "C" __global__
void DeviationKernel(double Mean, double *Data, uint64_t Count, double *Result)
{
  int Index = threadIdx.x + blockIdx.x * blockDim.x;
  int Stride = blockDim.x * gridDim.x;
  for(uint64_t i=Index; i<Count; i+=Stride)
  {
#if 0
    Result[i] = Data[i]-Mean;
#else
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
    Result[i] += Data[i]-Mean;
#endif
  }
  return;
}
extern "C" void SLRDeviationCuda(double Mean, double *Data, uint64_t Count, double *Result)
{
  double *DeviceDataSet;
  double *DeviceResult;
  // 記憶を割り当てコッピ
  uint64_t MemSize = sizeof(double)*Count;
  hipMalloc(&DeviceDataSet, MemSize);
  hipMalloc(&DeviceResult, MemSize);
  hipMemcpy(DeviceDataSet, Data, MemSize, hipMemcpyHostToDevice);
  
  hipError_t err;
  err = hipGetLastError(); // `hipGetLastError` will return the error from above.
  if (err != hipSuccess)
  {
    printf("Error: %s\n", hipGetErrorString(err));
  }
  // スレッドの準備
  int DeviceId = 0;
  hipGetDevice(&DeviceId);
  int SMCount = 0;
  hipDeviceGetAttribute(&SMCount, hipDeviceAttributeMultiprocessorCount, DeviceId);
  uint64_t ThreadCountPerBlock = SMCount;
  uint64_t BlockCount  = (Count + ThreadCountPerBlock - 1)/ThreadCountPerBlock;
  DeviationKernel<<<BlockCount, ThreadCountPerBlock>>>(Mean, DeviceDataSet, Count, DeviceResult);
  hipDeviceSynchronize();
  hipMemcpy(Result, DeviceResult, MemSize, hipMemcpyDeviceToHost);
  // 掃除
  hipFree(DeviceDataSet);
  hipFree(DeviceResult);
  return;
}
extern "C" void BrickSortCuda(brick_sorter *BrickSorter)
{
  //calculate thread count based on list size
  BrickSortKernel<<<BrickSorter->Count, 1>>>();
  //get device memory copy struct into gpu memory
  //get device memory copy struct data into gpu memory
  //pass in to kernel
  hipDeviceSynchronize();
  //copy everything back into host memory
  //return...
  return;
}
// int main(void) { return; } not needed