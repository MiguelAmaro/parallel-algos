#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "bricksorter.h"

extern "C" __global__
void CudaKernel(void)
{
  printf("hello world from cuda <%d,%d>\n",blockIdx.x, threadIdx.x);
  return;
}
extern "C" void CudaRunCodeFromC(void)
{
  CudaKernel<<<10, 1>>>();
  hipDeviceSynchronize();
  return;
}

extern "C" __global__
void BrickSortKernel(void)
{
  printf("hello from cuda brick sorter <%d,%d>\n",blockIdx.x, threadIdx.x);
  return;
}
extern "C" void BrickSortCuda(brick_sorter *BrickSorter)
{
  //calculate thread count based on list size
  BrickSortKernel<<<BrickSorter->Count, 1>>>();
  //get device memory copy struct into gpu memory
  //get device memory copy struct data into gpu memory
  //pass in to kernel
  hipDeviceSynchronize();
  //copy everything back into host memory
  //return...
  return;
}
// int main(void) { return; } not needed