
#include <hip/hip_runtime.h>
#include <stdio.h>
extern "C" __global__
void CudaKernel(void)
{
  printf("hello world from cuda <%d,%d>\n",blockIdx.x, threadIdx.x);
  return;
}
extern "C" void CudaRunCodeFromC(void)
{
  CudaKernel<<<10, 1>>>();
  hipDeviceSynchronize();
  return;
}
// int main(void) { return; } not needed